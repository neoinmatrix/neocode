// one dimension of mean filter designed and coded by neo 
/*
��Ŀ��
��ֵ�˲���
����һά������о�ֵ�˲���
����ÿ���������Ϊ���Դ˵�Ϊ���İ뾶Ϊr�����鵥Ԫ��
ȡֵ���ң���ƽ��ֵ���Թ��������в����������

Ҫ��:
1.C ����ʵ�ִ���
2.Cuda ����ʵ�ִ���
3.shared memory��ʹ��
4.���С���ݷ��ʲ���Խ��
5.��������ʱ��ͼ������
6.֧�ִ����ݵĴ���

thinking: 

the data of margin side can be dealed by this  (i-j+n)%n 
shaped the array  like circle 

in the same block ,the threads  visit the  data range in [r-i r r+i]
so copy global memory to shared memory to boost the speed

the shared memory is 48KB 
so the num of radius    3*r<= (48KB/4B)  =>  r <= 4K 

test data: 
10 3 1
100000 100 0
100000 50 0

*/
#include "hip/hip_runtime.h"

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <fstream>

#define  MIN(a,b) (a<b?a:b)
#define MAX_BLOCK 1024   //  biggest block numbers 
using namespace std;

//just with gpu  
__global__ void calcWithGPU_filter(float *b, const float *a, const int n,const int r){
	//__shared__ mean   we can add the shared memory to boost calc
	int g = blockIdx.x;
	int t = threadIdx.x;
	int i_global,i_inner;
	i_inner=g*r+t;                   //�����䵽 block �е� thread �� ����� �̶߳�Ӧ�� ��ֵ�˲����λ��
	if(i_inner>=n){                  //Խ�紦��
		return ;
	}
	float sum=0;
	for(int i=-r;i<=r;i++){         //������ֵ�˲��Ľ��  ȡ 2*r+1 �����ľ�ֵ
		i_global=(i_inner+i+n)%n;   //�Ҿ��Ŀ��� ȡģ ����ֵ�˲��� ��Ե���� ͷβ��� �����˲�
		sum+=a[i_global];
	}
	b[i_inner]=sum/(2*r+1);
}
 
//using the shared memory to save time 
__global__ void calcWithGPU_filter_shared(float *b, const float *a, const int n,const int r){
	//__shared__ mean   we can add the shared memory to boost calc
	int g = blockIdx.x;
	int t = threadIdx.x;
	int i_global,i_inner;
	extern __shared__ float cache[]; //dynamic shared memory allocation
	i_inner=g*r+t;  

	// copy ���� ��global �� �� shared memory ��  ÿ��λ����Ҫ�ԳƸ������� 
	i_global=(i_inner-r+n)%n; 
	cache[t]=a[i_global];
	i_global=(i_inner+n)%n; 
	cache[t+r]=a[i_global];
	i_global=(i_inner+r+n)%n; 
	cache[t+2*r]=a[i_global];

	__syncthreads();

	if(i_inner>=n){                            
		return ;
	}
	float sum=0;
	for(int i=-r;i<=r;i++){         //������ֵ�˲��Ľ��  ȡ 2*r+1 �����ľ�ֵ
		sum+=cache[r+t+i];
	}
	b[i_inner]=sum/(2*r+1);
}

//using the shared memory to save time 
__global__ void calcWithGPU_filter_shared_bd(float *b, const float *a, const int n,const int r){
	//__shared__ mean   we can add the shared memory to boost calc
	int g = blockIdx.x;
	int t = threadIdx.x;
	int i_global,i_inner;

	extern __shared__ float cache[]; //dynamic shared memory allocation
	
	int blocknum=r*MAX_BLOCK;    //the turns of  r*MAX_BLOCK 
	for(i_inner=g*r+t;i_inner<n+blocknum;i_inner=i_inner+blocknum){
		// copy ���� ��global �� �� shared memory ��  ÿ��λ����Ҫ�ԳƸ������� 
		i_global=(i_inner-r+n)%n; 
		cache[t]=a[i_global];
		i_global=(i_inner+n)%n; 
		cache[t+r]=a[i_global];
		i_global=(i_inner+r+n)%n; 
		cache[t+2*r]=a[i_global];

		__syncthreads();

		if(i_inner>=n){                            
			return ;
		}
		float sum=0;
		for(int i=-r;i<=r;i++){         //calc the sum of 2*r+1 data
			sum+=cache[r+t+i];
		}
		b[i_inner]=sum/(2*r+1);        // get the result of mean filter

		__syncthreads();
	}
	
}

//cpu mean filter process in detail
void calcWithCPU_filter(float *b, const float *a, const int n,const int r){
	if(r>n)       //can not calc
		return ;
	int i,j,index;
	float sum=0;
	for(i=0;i<n;i++){
		sum=0;
		for(j=-r;j<=r;j++){
			index=(i+j+n)%n;  //�Ҿ��Ŀ��� ȡģ ����ֵ�˲��� ��Ե���� ͷβ��� �����˲�
			sum+=a[index];
		}
		b[i]=sum/(2*r+1);
	}
}

//calc the error between cpu and gpu data
void calcErrorBetweenData(const float *b, const float *a, const int n,const int type){
	float error=0.0f;
	float tmp;
	ofstream file;
	if(type>0){
		if(type==1){
			file.open("result_nobd.txt");
			file.clear();
		}
		if(type==2){
			file.open("result_bd.txt");
			file.clear();
		}
	}
	for(int i=0;i<n;i++){
		tmp=b[i]-a[i];
		error+=tmp;
		if(type>0){
			file<<i<<" "<<b[i]<<" "<<a[i]<<endl;
		}
	}
	if(type>0){
		file.close();
	}
	printf(" the error between two data : %.3f \n",error);
}

//data to print
void print_data(float *b,  float *a, const int n,const int r){
	for(int i=0;i<n;i++){
		printf(" %f %f \n",a[i],b[i]);
	}
}

//cpu block to filter data
void process_cpu(float *b,  float *a, const int n,const int r, double *time,const bool print=true){
	double  duration;  
	clock_t begin, end;  
	begin = clock();  
	calcWithCPU_filter(b,a,n,r);
	end = clock();  
	duration = (double)( end -begin )*1000  / CLOCKS_PER_SEC;
	printf(" cpu mean_filter result: \n");
	(*time)=duration;
	if(print){
		printf(" [data] [filter] \n");
		print_data(b,a,n,r);
	}
	printf(" Time elapsed :  %3.3f ms \n", duration);
}

// gpu block to filter data   contains two branches  the filter without shared memory  or without shared memory
void process_gpu(float *b,  float *a, const int n,const int r ,int type, double *time,const bool print=true){
	float *dev_a,*dev_b;
	hipEvent_t     start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start, 0 );

	hipMalloc((void**)&dev_a, n * sizeof(float));
	hipMalloc((void**)&dev_b, n * sizeof(float));
	hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice);
	if(type==1)
		calcWithGPU_filter<<<MIN(((n+r-1)/r),MAX_BLOCK),r>>>(dev_b, dev_a, n, r);
	if(type==2)
		calcWithGPU_filter_shared<<<MIN(((n+r-1)/r),MAX_BLOCK),r,r*3*sizeof(float)>>>(dev_b, dev_a, n, r);
	if(type==3)
		calcWithGPU_filter_shared_bd<<<MIN(((n+r-1)/r),MAX_BLOCK),r,r*3*sizeof(float)>>>(dev_b, dev_a, n, r);

	hipMemcpy(b, dev_b, n * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(dev_a);
	hipFree(dev_b);
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	float   elapsedTime;
	hipEventElapsedTime( &elapsedTime, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
	printf(" cuda mean_filter result: \n");
	(*time)=elapsedTime;
	if(print){
		printf(" [data] [filter] \n");
		print_data(b,a,n,r);
	}
	printf( " Time elapsed :  %3.3f ms \n", elapsedTime );
}

int main(){
	int n,r,debug_status;
	bool debug=false;
	printf("input the n and r [or is_debug(1)]to filter data : ");
	scanf(" %d %d %d",&n,&r,&debug_status);                           //n sizes of array      r the radius of filter
	float *a,*b,*c,*d,*e;
	double time_cpu,time_gpu_1,time_gpu_2,time_gpu_3;
	a=(float*)malloc(sizeof(float)*n);
	b=(float*)malloc(sizeof(float)*n);
	c=(float*)malloc(sizeof(float)*n);
	d=(float*)malloc(sizeof(float)*n);
	e=(float*)malloc(sizeof(float)*n);

	//srand((unsigned)time(NULL));/*������*/
	for(int i=0;i<n;i++){ 
		if(debug_status>0){
			a[i]=(float)i;
		}else{
			a[i]=rand()%RAND_MAX %100 ;
		}
		//a[i]=powf(a[i],7);
	}
	debug=(debug_status>0)?true:false;
	//CPU===============================================================
	process_cpu(b,a,n,r,&time_cpu,debug);
	//GPU===============================================================
	process_gpu(c,a,n,r,1,&time_gpu_1,debug);     // without shared_memory 
	process_gpu(d,a,n,r,2,&time_gpu_2,debug);     // with shared_memory
	process_gpu(e,a,n,r,3,&time_gpu_3,debug);     // with shared_memory big data
	//printf( "\n  %f %f %f \n",time_cpu,time_gpu_1,time_gpu_2);

	//printf( "\n the error: [ cpu mean filter data / cuda mf data without bd ] \n");
	//calcErrorBetweenData(b,d,n);
	printf( "\n the error: [ cpu mean filter data / cuda mf data without bd] \n");
	calcErrorBetweenData(b,d,n,1);
	printf( "\n the error: [ cpu mean filter data / cuda mf data with bd] \n");
	calcErrorBetweenData(b,e,n,2);

	printf("\n speedup rate:\n");
	if(time_cpu>time_gpu_1 &&time_cpu!=0&&time_gpu_1!=0){
		printf( " cuda without shared memory => speedup rate :  %d:1  \n", (int)ceil(time_cpu/time_gpu_1 ));
	}else{
		printf( " cuda without shared memory => no speed up \n");
	}

	if(time_cpu>time_gpu_2 &&time_cpu!=0&&time_gpu_2!=0){
		printf( " cuda with shared memory => speedup rate :  %d:1  \n", (int)ceil(time_cpu/time_gpu_3 ));
	}else{
		printf( " cuda with shared memory => no speed up \n");
	}
	
	if(time_gpu_1>time_gpu_2 &&time_gpu_1!=0&&time_gpu_2!=0){
		printf( " cuda without shared memory / cuda with shared memory => speedup rate :  %d:1  \n", (int)ceil(time_gpu_1/time_gpu_3 ));
	}else{
		printf( " cuda without shared memory / cuda with shared memory => no speed up \n");
	}
	
	
	return 0;
}
